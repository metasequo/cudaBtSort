#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <limits.h>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline.h>
#include <cutil_inline_runtime.h>

// �v�f���A�u���b�N�T�C�Y�A���[�v��
#define SIZE 4096
#define BLOCK_SIZE 256
#define LOOP 10000

__global__
static void BtSort(int* inData);

int main(){

	// �ϐ��錾
	int* targetData;
	int i, j;

	// �������m��
	targetData = (int*)malloc(sizeof(int) * SIZE);

	// �f�o�C�X���̕ϐ��錾
	int* dTargetData;

	// �f�o�C�X�������m��
	cutilSafeCall(hipMalloc((void**)&dTargetData, sizeof(int) * SIZE));
	cutilSafeCall(hipMemcpy(dTargetData, targetData, sizeof(int) * SIZE, hipMemcpyHostToDevice));

	// �u���b�N�T�C�Y�A�O���b�h�T�C�Y�ݒ�
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(SIZE / BLOCK_SIZE, SIZE / BLOCK_SIZE, SIZE / BLOCK_SIZE);
//	if (SIZE / BLOCK_SIZE < 1)	dim3 grid(1);

	// �^�C�}�[�ϐ��̐錾�A����J�n
	printf("Bitonic sort start in the GPU!\n");
	printf("Element count\t:\t%d\n", SIZE);
	printf("BlockSize.X\t:\t%d\nBlockSize.Y\t:\t%d\nBlockSize.Z\t:\t%d\n", block.x, block.y, block.y);
	printf("GridSize.X\t:\t%d\nGridSize.Y\t:\t%d\nGridSize.X\t:\t%d\n", grid.x, grid.y, grid.z);
	printf("Loop count\t:\t%d\n", LOOP);
	float millseconds = 0.0f, sum = 0.0f, ave = 0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// ���C�����[�v
	for (int k = 0; k < 10; k++){
		sum = 0.0f;
		for (i = 0; i < LOOP; i++){
			// �v�f��������
			for (j = 0; j < SIZE; j++)
				targetData[j] = (int) ((rand() / ((double) RAND_MAX + 0.1f))* INT_MAX);
				
			// �L�^�J�n�A�J�[�l���֐����s
			hipEventRecord(start, 0);
			BtSort <<<grid, block>>>(dTargetData);
			hipDeviceSynchronize();
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&millseconds, start, stop);
			sum += millseconds;
		}
		printf("Time required\t:\t%f minutes\n", sum / 1000);
		ave += sum;
	}

	// ����I��
//	printf("Element count\t:\t%d\n", SIZE);
//	printf("BlockSize\t:\t%d\nGridSize\t:\t%d\n", BLOCK_SIZE, SIZE / BLOCK_SIZE);
//	printf("Loop count\t:\t%d\n", LOOP);
	printf("Time average\t:\t%f minutes\n", ave /10000);

	// ���ʂ̗̈�̊m�ۂƁA�f�o�C�X������̃������]��
	cutilSafeCall(hipMemcpy(targetData, dTargetData, sizeof(int) * SIZE, hipMemcpyDeviceToHost));

	// ���������
	free(targetData);
	cutilSafeCall(hipFree(dTargetData));

	hipDeviceReset();
}

// �o�C�g�j�b�N�\�[�g����J�[�l���֐�
__global__
static void BtSort(int* inData){
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// �O���̓}�[�W�A�����͕��������̃��[�v
	for(unsigned int length = 2; length <= SIZE; length *= 2){
		for(unsigned int mlength = length / 2; mlength > 0; mlength /= 2){
			unsigned int ixj = idx ^ mlength;

			if(ixj > idx){
				int tmp;

				// �������~�������f���ē���ւ�
				if((idx & ixj) == 0){
					if(inData[idx] > inData[ixj]){
						tmp = inData[ixj];
						inData[ixj] = inData[idx];
						inData[idx] = tmp;
					}
				}else{
					if(inData[idx] < inData[ixj]){
						tmp = inData[ixj];
						inData[ixj] = inData[idx];
						inData[idx] = tmp;
					}
				}
			}
			__threadfence();	//�A�N�Z�X�\�܂őҋ@
			__syncthreads();	//�X���b�h����
		}
	}
}